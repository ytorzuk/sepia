#include "hip/hip_runtime.h"
#include "sepia.h"

#define BLOCK_SIZE (32)

namespace gpu
{

__global__ void sepiaKernel(unsigned char * image, unsigned width, unsigned height)
{
        int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
        int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

        int cIdx = ( row * width + col ) * 3;

        unsigned char b = *( image + cIdx     );
        unsigned char g = *( image + cIdx + 1 );
        unsigned char r = *( image + cIdx + 2 );

        *( image + cIdx     ) = .274f * r + .134f * g + .192f * b; // Red channel
        *( image + cIdx + 1 ) = .346f * r + .486f * g + .168f * b; // Green channel
        *( image + cIdx + 2 ) = .242f * r + .669f * g + .089f * b; // Blue channel
}


void applySepia(const unsigned char * inBuffer, unsigned width, unsigned height, unsigned char * outBuffer)
{
    unsigned char * dImage;
    unsigned size = width * height * 3;
    hipMalloc(&dImage, size);
    hipMemcpy(dImage, inBuffer, size, hipMemcpyHostToDevice);

    dim3 dimBlock( BLOCK_SIZE, BLOCK_SIZE );
    dim3 dimGrid( width / dimBlock.x, height / dimBlock.y );
    sepiaKernel<<<dimGrid, dimBlock>>>(dImage, width, height);

    hipMemcpy(outBuffer, dImage, size, hipMemcpyDeviceToHost);
    hipFree(dImage);
}
}

